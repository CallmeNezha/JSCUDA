#include "hip/hip_runtime.h"
#define  JSCUDA_DLL_EXPORT

#include "cuda_common_kernel.cuh"
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#ifdef  JSCUDA_DLL_EXPORT
#define JSCUDA_DLL_API  __declspec(dllexport)
#else
#define JSCUDA_DLL_API  __declspec(dllimport)
#endif

extern "C"
{

    JSCUDA_DLL_API 
    void cudaDeviceInit(int argc, char **argv)
    {
        int devID;
        // use command - line specified CUDA device, otherwise use device with highest Gflops / s
        devID = findCudaDevice(argc, (const char **)argv);
        if (devID < 0)
        {
            printf("No CUDA Capable devices found, exiting...\n");
            exit(EXIT_SUCCESS);
        }
        hipDeviceProp_t deviceProp;
        checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
        U_NUM_BLOCKSIZE = (deviceProp.major < 2) ? 16 : 32;
        U_NUM_THREAD = U_NUM_BLOCKSIZE * U_NUM_BLOCKSIZE;
    }
    JSCUDA_DLL_API
    void cudaDeviceReset_t()
    {
        checkCudaErrors(hipDeviceReset());
    }
    JSCUDA_DLL_API
    void cudaMalloc_t(void **devPtr, size_t size)
    {
        checkCudaErrors(hipMalloc(devPtr, size));
    }
    JSCUDA_DLL_API
    void cudaFree_t(void *devPtr)
    {
        checkCudaErrors(hipFree(devPtr));
    }
    JSCUDA_DLL_API
    void cudaSync()
    {
        checkCudaErrors(hipDeviceSynchronize());
    }
    JSCUDA_DLL_API
    void cudaMemcpyHostToDevice_t(const void *host, void *device, int offset, size_t size)
    {
        checkCudaErrors(hipMemcpy((char *)device + offset, host, size, hipMemcpyHostToDevice));
    }
    JSCUDA_DLL_API
    void cudaMemcpyDeviceToDevice_t(void *dst, const void *src, int offset, size_t size)
    {
        checkCudaErrors(hipMemcpy((char *)dst + offset, src, size, hipMemcpyDeviceToDevice));
    }
    JSCUDA_DLL_API
    void cudaMemcpyDeviceToHost_t(const void *device, void *host, int offset, size_t size)
    {
        checkCudaErrors(hipMemcpy((char *)host + offset, device, size, hipMemcpyDeviceToHost));
    }

    //Round a / b to nearest higher integer value
    inline uint iDivUp(uint a, uint b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }
    // compute grid and thread block size for a given number of elements
    void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }
    JSCUDA_DLL_API
    void vectorAdd(const float* v1d, const float* v2d, float* vout, uint size)
    {
        uint numBlocks, numThreads;
        computeGridSize(size, U_NUM_THREAD, numBlocks, numThreads);
        vectorAddD << <numBlocks, numThreads >> >(v1d, v2d, vout, size);
        getLastCudaError("Kernel execution failed");
    }

    // global memory write operation in queue maybe not faster than CPU method
    JSCUDA_DLL_API
    float vectorInnerProduct(const float* v1d, const float* v2d, uint size)
    {
        float vout = 0.f;
        for (uint i = 0; i < size; i++)
        {
            vout += v1d[i] * v2d[i];
        }
        return vout;
    }

    JSCUDA_DLL_API
    void cublasCreate_t(hipblasHandle_t *handle)
    {
        checkCudaErrors(hipblasCreate(handle));
    }

    JSCUDA_DLL_API
    void cublasDestroy_t(hipblasHandle_t handle)
    {
        checkCudaErrors(hipblasDestroy(handle));
    }



    JSCUDA_DLL_API
    ErrorType matrixMulMatrix_blas(const hipblasHandle_t handle, const Matrix& matAd, const Matrix& matBd, Matrix& matOd)
    {
        if (matAd.numCol != matBd.numRow) return JC_PARAM_ERROR;
        if (matAd.numRow != matOd.numRow || matBd.numCol != matOd.numCol) return JC_PARAM_ERROR;

        dim3 threads(U_NUM_BLOCKSIZE, U_NUM_BLOCKSIZE);
        dim3 grid(ceilfuint((float)matOd.numCol / threads.x), ceilfuint((float)matOd.numRow / threads.y));
        const float alpha = 1.f;
        const float beta = 0.f;
        checkCudaErrors(hipblasSgemm(handle
            , HIPBLAS_OP_N
            , HIPBLAS_OP_N
            , matAd.numRow
            , matBd.numCol
            , matAd.numCol
            , &alpha
            , matAd.elements
            , matAd.numRow
            , matBd.elements
            , matBd.numRow
            , &beta
            , matOd.elements
            , matOd.numRow));

        getLastCudaError("Kernel execution failed");
        return JC_SUCCESS;
    }
}