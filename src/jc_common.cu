#define  JSCUDA_DLL_EXPORT

#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#ifdef  JSCUDA_DLL_EXPORT
#define JSCUDA_DLL_API  __declspec(dllexport)
#else
#define JSCUDA_DLL_API  __declspec(dllimport)
#endif

extern "C"
{
    //Default parameter, may scalar by 2 for Fermi or above
    unsigned int  U_NUM_THREAD = 256;
    unsigned int  U_NUM_BLOCKSIZE = 16;

    JSCUDA_DLL_API 
    bool cudaDeviceInit(int argc, char **argv)
    {
        int devID;
        // use command - line specified CUDA device, otherwise use device with highest Gflops / s
        devID = findCudaDevice(argc, (const char **)argv);
        if (devID < 0)
        {
            printf("No CUDA Capable devices found, exiting...\n");
            return false;
        }
        hipDeviceProp_t deviceProp;
        checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
        U_NUM_BLOCKSIZE = (deviceProp.major < 2) ? 16 : 32;
        U_NUM_THREAD = U_NUM_BLOCKSIZE * U_NUM_BLOCKSIZE;
        return true;
    }
    JSCUDA_DLL_API
    void cudaDeviceReset_t()
    {
        checkCudaErrors(hipDeviceReset());
    }
    JSCUDA_DLL_API
    void cudaMalloc_t(void **devPtr, size_t size)
    {
        checkCudaErrors(hipMalloc(devPtr, size));
    }
    JSCUDA_DLL_API
    void cudaFree_t(void *devPtr)
    {
        checkCudaErrors(hipFree(devPtr));
    }
    JSCUDA_DLL_API
    void cudaSync()
    {
        checkCudaErrors(hipDeviceSynchronize());
    }
    JSCUDA_DLL_API
    void cudaMemcpyHostToDevice_t(const void *host, void *device, int offset_h, int offset_d, size_t size)
    {
        checkCudaErrors(hipMemcpy((char *)device + offset_d, (char *)host + offset_h, size, hipMemcpyHostToDevice));
    }
    JSCUDA_DLL_API
    void cudaMemcpyDeviceToDevice_t(void *dst, const void *src, int offset_d, int offset_s, size_t size)
    {
        checkCudaErrors(hipMemcpy((char *)dst + offset_d, (char *)src + offset_s, size, hipMemcpyDeviceToDevice));
    }
    JSCUDA_DLL_API
    void cudaMemcpyDeviceToHost_t(const void *device, void *host, int offset_d, int offset_h, size_t size)
    {
        checkCudaErrors(hipMemcpy((char *)host + offset_h, (char *)device + offset_d, size, hipMemcpyDeviceToHost));
    }

    
}